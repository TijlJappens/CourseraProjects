
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// CUDA kernel to add two vectors
__global__ void vectorAddition(const float* A, const float* B, float* C, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int numElements = 1 << 20; // 2^20 elements

    // Allocate memory for host vectors
    float *h_A = new float[numElements];
    float *h_B = new float[numElements];
    float *h_C = new float[numElements];

    // Initialize host vectors with some values
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate memory for device vectors
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, numElements * sizeof(float));
    hipMalloc((void**)&d_B, numElements * sizeof(float));
    hipMalloc((void**)&d_C, numElements * sizeof(float));

    // Copy host vectors to device
    hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size
    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;

    // Launch the vector addition kernel
    vectorAddition<<<gridSize, blockSize>>>(d_A, d_B, d_C, numElements);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

    // Check for errors in kernel launch or execution
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(hipError_t) << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    std::cout << "Reached end of program." << std::endl;
    // Do something with the result if needed

    return 0;
}
